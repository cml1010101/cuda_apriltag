#include "hip/hip_runtime.h"
#include <apriltag_math.h>
__device__
void device_mat33_chol(const double* A, double* R)
{
    // A[0] = R[0]*R[0]
    R[0] = sqrt(A[0]);

    // A[1] = R[0]*R[3];
    R[3] = A[1] / R[0];

    // A[2] = R[0]*R[6];
    R[6] = A[2] / R[0];

    // A[4] = R[3]*R[3] + R[4]*R[4]
    R[4] = sqrt(A[4] - R[3]*R[3]);

    // A[5] = R[3]*R[6] + R[4]*R[7]
    R[7] = (A[5] - R[3]*R[6]) / R[4];

    // A[8] = R[6]*R[6] + R[7]*R[7] + R[8]*R[8]
    R[8] = sqrt(A[8] - R[6]*R[6] - R[7]*R[7]);

    R[1] = 0;
    R[2] = 0;
    R[5] = 0;
}
__global__
void kernel_mat33_chol(const double* A, double* R)
{
    device_mat33_chol(A, R);
}
void mat33_chol(const double* A, double* R)
{
    kernel_mat33_chol<<<1, 1>>>(A, R);
}
__device__
void device_mat33_lower_tri_inv(const double* A, double* R)
{
    // A[1] = R[0]*R[3];
    R[3] = A[1] / R[0];

    // A[2] = R[0]*R[6];
    R[6] = A[2] / R[0];

    // A[4] = R[3]*R[3] + R[4]*R[4]
    R[4] = sqrt(A[4] - R[3]*R[3]);

    // A[5] = R[3]*R[6] + R[4]*R[7]
    R[7] = (A[5] - R[3]*R[6]) / R[4];

    // A[8] = R[6]*R[6] + R[7]*R[7] + R[8]*R[8]
    R[8] = sqrt(A[8] - R[6]*R[6] - R[7]*R[7]);

    R[1] = 0;
    R[2] = 0;
    R[5] = 0;
}
__global__
void kernel_mat33_lower_tri_inv(const double* A, double* R)
{
    device_mat33_lower_tri_inv(A, R);
}
void mat33_lower_tri_inv(const double* A, double* R)
{
    kernel_mat33_lower_tri_inv<<<1, 1>>>(A, R);
}
__device__
void device_mat33_sym_solve(const double* A, const double* B, double* R)
{
    double L[9];
    device_mat33_chol(A, L);

    double M[9];
    device_mat33_lower_tri_inv(L, M);

    double tmp[3];
    tmp[0] = M[0]*B[0];
    tmp[1] = M[3]*B[0] + M[4]*B[1];
    tmp[2] = M[6]*B[0] + M[7]*B[1] + M[8]*B[2];

    R[0] = M[0]*tmp[0] + M[3]*tmp[1] + M[6]*tmp[2];
    R[1] = M[4]*tmp[1] + M[7]*tmp[2];
    R[2] = M[8]*tmp[2];
}
__global__
void kernel_mat33_sym_solve(const double* A, const double* B, double* R)
{
    device_mat33_sym_solve(A, B, R);
}
void mat33_sym_solve(const double* A, const double* B, double* R)
{
    kernel_mat33_sym_solve<<<1, 1>>>(A, B, R);
}